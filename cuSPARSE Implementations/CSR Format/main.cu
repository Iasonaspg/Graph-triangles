/**********************************************************************
 *
 * readCSV.c -- readCSV function for reading the input from
 *              a .csv file
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include <ctype.h>  
 #include <inttypes.h>
 #include <errno.h>
 #include <math.h>
 #include "readCSV.h"
 #include "validation.h"

 

int main(int argc, char** argv){
 
    char* fName = argv[1];
    int N, M, nT_Mat;
    double matlab_time;

    csrFormat A, C;

    printf("Reading of dataset and validation file has started\n");
    readCSV(fName, &A, &N, &M, &nT_Mat, &matlab_time);
    printf("Reading of dataset and validation file has ended\n");

    printf("Nonzeros = %d\n", A.nnz);
 
    printf("Validation File:\n N = %d, M = %d\n Matlab result was %d, produced in %lf\n", N, M, nT_Mat, matlab_time);

    // Multiplication of two sparse arrays using cuSparse
    mulSparse(&A,&C,N);

    // Move adjancy array to gpu memory
    float* devVal;
    int* devCol, *devRow;
    int nnzA = A.nnz;
    CHECK(hipMalloc((void**)&devVal,nnzA*sizeof(float)));
    CHECK(hipMalloc((void**)&devCol,nnzA*sizeof(int)));
    CHECK(hipMalloc((void**)&devRow,(N+1)*sizeof(int)));
    CHECK(hipMemcpy(devVal,A.csrVal,nnzA*sizeof(float),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(devCol,A.csrColInd,nnzA*sizeof(int),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(devRow,A.csrRowPtr,(N+1)*sizeof(int),hipMemcpyHostToDevice));

    // Use a struct to access it easier
    csrFormat B;
    B.csrColInd = devCol;
    B.csrRowPtr = devRow;
    B.csrVal = devVal;
    B.nnz = A.nnz;

    // Initialize value of triangles to 0
    int* sum, *h_sum;
    CHECK(hipMalloc((void**)&sum,sizeof(int)));
    h_sum = (int*)malloc(sizeof(int));
    *h_sum = 0;
    CHECK(hipMemcpy(sum,h_sum,sizeof(int),hipMemcpyHostToDevice));
    
        
    // double st1 = cpuSecond();
    //filter<<<160,1024>>>(B,C,counter1,counter2);
    // CHECK(hipPeekAtLastError());
    // CHECK(hipDeviceSynchronize());
    //printf("Time filtering on GPU: %lf sec\n",cpuSecond()-st1);
    //printf("Vrhka tosa simeia: %d kai midenisa tosa: %d\n",*counter1,*counter2);

    // double st2 = cpuSecond();
    // findTriangles<<<160,1024>>>(B,C,sum,counter);
    // CHECK(hipPeekAtLastError());
    // CHECK(hipDeviceSynchronize());
    // printf("Time on GPU: %lf sec\n",cpuSecond()-st2);

    
    // Call our kernel to find the number of triangles
    double st3 = cpuSecond();
    findTriangles<<<20,1024>>>(B,C,sum,N);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    hipMemcpy(h_sum,sum,sizeof(int),hipMemcpyDeviceToHost);
    printf("Triangles using CSR format: %d\n",h_sum[0]/3);
    //printf("Triangles naive: %d\n",counter[0]/6);
    printf("Time on GPU using CSR format: %lf sec\n",cpuSecond()-st3);

    if (validation(h_sum[0]/3,nT_Mat)){
        printf("Validation on GPU: PASSED\n");
    }
    

    free(A.csrVal);
    free(A.csrColInd);
    free(A.csrRowPtr);
    free(h_sum);
    hipFree(devCol);
    hipFree(devRow);
    hipFree(devVal);
    hipFree(C.csrVal);
    hipFree(C.csrColInd);
    hipFree(C.csrRowPtr);
    hipFree(sum);
    
    return 0; 
}