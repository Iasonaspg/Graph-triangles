#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <sys/time.h>
#include "readCSV.h"


double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


// int main()
// {
//     // Initialize cuSPARSE
//     hipsparseHandle_t handle;   
//     hipsparseCreate(&handle);

//     const int N = 3;                // Number of rows and columns

//     // Host side dense matrices
//     float *h_A_dense = (float*)malloc(N * N * sizeof(float));
//     float *h_x_dense = (float*)malloc(N *     sizeof(float));
//     float *h_y_dense = (float*)malloc(N *     sizeof(float));

//     // Column-major ordering
//     h_A_dense[0] = 1;  h_A_dense[4] = 6;     h_A_dense[8]  = 5; 
//     h_A_dense[1] = 0; h_A_dense[5] = 0;      
//     h_A_dense[2] = 1;  h_A_dense[6] = 4;       
//     h_A_dense[3] = 0;      h_A_dense[7] = 0.0;          

//     // Initializing the data and result vectors
//     for (int k = 0; k < N; k++) {
//         h_x_dense[k] = 1.;
//         h_y_dense[k] = 0.;
//     }

//     // Create device arrays and copy host arrays to them
//     float *d_A_dense;  
//     hipMalloc(&d_A_dense, N*N*sizeof(float));
//     float *d_x_dense;  
//     hipMalloc(&d_x_dense, N*sizeof(float));
//     float *d_y_dense;  
//     hipMalloc(&d_y_dense, N*sizeof(float));
//     hipMemcpy(d_A_dense, h_A_dense, N*N*sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(d_x_dense, h_x_dense, N*sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(d_y_dense, h_y_dense, N*sizeof(float), hipMemcpyHostToDevice);


//     // Descriptor for sparse matrix A and C
//     hipsparseMatDescr_t descrA,descrC;     
//     hipsparseCreateMatDescr(&descrA);
//     hipsparseCreateMatDescr(&descrC);
//     //hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
//     //hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);  

//     int nnzA = 0;                           // Number of nonzero elements in dense matrix A

//     const int lda = N;                      // Leading dimension of dense matrix

//     // Device side number of nonzero elements per row of matrix A
//     int *d_nnzPerVectorA;   
//     hipMalloc(&d_nnzPerVectorA, N * sizeof(*d_nnzPerVectorA));
//     hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, N, N, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA);

//     // Sparse matrix
//     float *d_A;            
//     hipMallocManaged(&d_A, nnzA*sizeof(*d_A));
//     int *d_A_RowIndices;    
//     hipMallocManaged(&d_A_RowIndices, (N + 1) * sizeof(*d_A_RowIndices));
//     int *d_A_ColIndices;    
//     hipMallocManaged(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices));

//     hipsparseSdense2csr(handle, N, N, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices);

//     cooFormat* test = (cooFormat*)malloc(sizeof(cooFormat));
//     test->nnz = nnzA;
//     test->cooValA = d_A;
//     test->cooColIndA = d_A_ColIndices;
//     test->cooRowIndA = d_A_RowIndices;
    
//     // for (int i=0;i<N+1;i++){
//     //     printf("Row: %d\n",d_A_RowIndices[i]);
//     // }
//     // for (int i=0;i<nnzA;i++){
//     //     printf("Col: %d\n",d_A_ColIndices[i]);
//     // }

//     //printf("Random value: %f\n",d_A[3]);
//     cooFormat* test1 = (cooFormat*)malloc(sizeof(cooFormat));
//     // hipMallocManaged(&test1,sizeof(csrFormat));

//     mulSparse(test,test1,N);
    

//     for (int i=0; i<test1->nnz;i++){
//         // printf("Values: %f\n",test1->csrVal[i]);
//     }
    
//     // const float alpha = 1.;
//     // const float beta  = 0.;
//     // hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nnzA, &alpha, descrA, d_A, d_A_RowIndices, d_A_ColIndices, d_x_dense,&beta, d_y_dense);

//     // hipMemcpy(h_y_dense,d_y_dense,N*sizeof(double),hipMemcpyDeviceToHost);

//     // printf("\nResult vector\n\n");
//     // for (int i = 0; i < N; ++i){
//     //     printf("h_y[%i] = %f\n", i, h_y_dense[i]);
//     // }


//     return 0;
// }

void mulSparse(cooFormat* A, cooFormat* C, int N){

    // Initialize cuSPARSE
    hipsparseHandle_t handle;   
    hipsparseCreate(&handle);

    int nnzA = A->nnz;

    float* devVal;
    int* devCol, *devRow;
    hipMallocManaged(&devVal,nnzA*sizeof(float));
    hipMallocManaged(&devCol,nnzA*sizeof(int));
    hipMallocManaged(&devRow,nnzA*sizeof(int));
    hipMemcpy(devVal,A->cooValA,nnzA*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(devCol,A->cooColIndA,nnzA*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devRow,A->cooRowIndA,nnzA*sizeof(int),hipMemcpyHostToDevice);

    int* csrRowPtrA;
    hipMallocManaged(&csrRowPtrA,(N+1)*sizeof(int));
    // hipMemcpy(csrRowPtrA,A->cooRowIndA,(N+1)*sizeof(int),hipMemcpyHostToDevice);
    hipsparseXcoo2csr(handle,devRow,nnzA,N,csrRowPtrA,HIPSPARSE_INDEX_BASE_ZERO);
    

    for (int i=0;i<N+1;i++){
        // printf("csrValA from func: %f\n",devVal[i]);
        // printf("csrRowPtrA from func: %d\n",csrRowPtrA[i]);
        //printf("csrColIndA from func: %d\n",devCol[i]);
    }
    for (int i=0;i<nnzA;i++){
        // printf("csrColIndA from func: %d for value: %f\n",devCol[i],devVal[i]);
    }

    // printf("Inside func: %f\n",A->csrVal[4]);

    // Descriptor for sparse matrix A and C
    hipsparseMatDescr_t descrA,descrC;     
    hipsparseCreateMatDescr(&descrA);
    hipsparseCreateMatDescr(&descrC);

    int* d_C_RowPtr;
    hipMallocManaged(&d_C_RowPtr,(N+1)*sizeof(*d_C_RowPtr));

    int nnzC;
    //hipMallocManaged(&nnzC,sizeof(int));
    hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseXcsrgemmNnz(handle,transA,transA,N,N,N,descrA,nnzA,csrRowPtrA,devCol,descrA,nnzA,csrRowPtrA,devCol,descrC,d_C_RowPtr,&nnzC);


    printf("Non zero of C: %d\n",nnzC);

    float* d_C;
    int *d_C_ColIndices;
    hipMallocManaged(&d_C, (nnzC)*sizeof(*d_C));
    hipMallocManaged(&d_C_ColIndices, (nnzC) * sizeof(*d_C_ColIndices));
    double start = cpuSecond();
    hipsparseScsrgemm(handle,transA,transA,N,N,N,descrA,nnzA,devVal,csrRowPtrA,devCol,descrA,nnzA,devVal,csrRowPtrA,devCol,descrC,d_C,d_C_RowPtr,d_C_ColIndices);
    printf("Time elapsed for multiplication: %f seconds\n",cpuSecond()-start);

    int* cooRowC;
    hipMallocManaged(&cooRowC,(nnzC)*sizeof(int));

    hipsparseXcsr2coo(handle,d_C_RowPtr,(nnzC),N,cooRowC,HIPSPARSE_INDEX_BASE_ZERO);

    printf("timi: %d\n",(nnzC));
    C->nnz = nnzC;
    C->cooValA = d_C;
    C->cooRowIndA = cooRowC;
    C->cooColIndA =  d_C_ColIndices;
}