/**********************************************************************
 *
 * readCSV.c -- readCSV function for reading the input from
 *              a .csv file
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include <ctype.h>  
 #include <inttypes.h>
 #include <errno.h>
 #include <math.h>
 #include "readCSV.h"

 

int main(int argc, char** argv){
 
    char* fName = argv[1];
    int N, M, nT_Mat;
    double matlab_time;

    csrFormat A, C;

    readCSV(fName, &A, &N, &M, &nT_Mat, &matlab_time);

    
    printf("Nonzeros = %d\n", A.nnz);
    

    printf("Validation File:\n N = %d, M = %d\n Matlab result was %d, produced in %lf\n", N, M, nT_Mat, matlab_time);

    mulSparse(&A,&C,N);

    float* devVal;
    int* devCol, *devRow;
    int nnzA = A.nnz;
    hipMallocManaged(&devVal,nnzA*sizeof(float));
    hipMallocManaged(&devCol,nnzA*sizeof(int));
    hipMallocManaged(&devRow,(N+1)*sizeof(int));
    hipMemcpy(devVal,A.csrVal,nnzA*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(devCol,A.csrColInd,nnzA*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devRow,A.csrRowPtr,(N+1)*sizeof(int),hipMemcpyHostToDevice);

    csrFormat B;
    B.csrColInd = devCol;
    B.csrRowPtr = devRow;
    B.csrVal = devVal;
    B.nnz = A.nnz;

    int* sum, *counter, *counter1, *counter2;
    hipMallocManaged(&sum,sizeof(int));
    hipMallocManaged((void**)&counter,sizeof(int));
    hipMallocManaged(&counter1,sizeof(int));
    hipMallocManaged(&counter2,sizeof(int));

    double st1 = cpuSecond();
    //filter<<<160,1024>>>(B,C,counter1,counter2);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    //printf("Time filtering on GPU: %lf sec\n",cpuSecond()-st1);
    //printf("Vrhka tosa simeia: %d kai midenisa tosa: %d\n",*counter1,*counter2);

    double st2 = cpuSecond();
    //findTrianglesShared<<<160,1024>>>(B,C,sum,counter);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    //printf("Time on GPU using shared memory: %lf sec\n",cpuSecond()-st2);

    double st3 = cpuSecond();
    *sum = 0;
    *counter = 0;
    //findTrianglesSum<<<160,1024>>>(B,C,sum,counter);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    //printf("Triangles: %d\n",sum[0]/6);
    //printf("Triangles naive: %d\n",counter[0]/6);
    //printf("Time on GPU using memory: %lf sec\n",cpuSecond()-st3);
    
    double st = cpuSecond();
    //findTrianglesCPU(&B,&C);
    //printf("Time on CPU: %lf sec\n",cpuSecond()-st);

    // for (int i=0;i<9;i++){
    // printf("Sample: %f\n",C.cooValA[i]);
    // }
    
    return 0; 
}