#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <sys/time.h>
#include "readCSV.h"


double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void mulSparse(csrFormat* A, csrFormat* C, int N){

    // Initialize cuSPARSE
    hipsparseHandle_t handle;   
    hipsparseCreate(&handle);

    int nnzA = A->nnz;

    // Allocate gpu memory to hold matrix in CSR format
    float* devVal;
    int* devCol, *devRowPtr;
    hipMalloc((void**)&devVal,nnzA*sizeof(float));
    hipMalloc((void**)&devCol,nnzA*sizeof(int));
    hipMalloc((void**)&devRowPtr,(N+1)*sizeof(int));
    hipMemcpy(devVal,A->csrVal,nnzA*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(devCol,A->csrColInd,nnzA*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devRowPtr,A->csrRowPtr,(N+1)*sizeof(int),hipMemcpyHostToDevice);


    // Descriptor for sparse matrix A and C
    hipsparseMatDescr_t descrA,descrC;     
    hipsparseCreateMatDescr(&descrA);
    hipsparseCreateMatDescr(&descrC);

    // Row pointer vector for the product matrix in CSR format
    int* d_C_RowPtr;
    hipMallocManaged(&d_C_RowPtr,(N+1)*sizeof(*d_C_RowPtr));

    // Calculate the number of C's non-zero values
    int nnzC;
    hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseXcsrgemmNnz(handle,transA,transA,N,N,N,descrA,nnzA,devRowPtr,devCol,descrA,nnzA,devRowPtr,devCol,descrC,d_C_RowPtr,&nnzC);

    printf("Non zero of C: %d\n",nnzC);


    // Allocate gpu memory to hold matrix C in CSR format
    float* d_C;
    int *d_C_ColIndices;
    hipMalloc((void**)&d_C, (nnzC)*sizeof(*d_C));
    hipMalloc((void**)&d_C_ColIndices, (nnzC) * sizeof(*d_C_ColIndices));
    
    // Calculate sparse array C
    double start = cpuSecond();
    hipsparseScsrgemm(handle,transA,transA,N,N,N,descrA,nnzA,devVal,devRowPtr,devCol,descrA,nnzA,devVal,devRowPtr,devCol,descrC,d_C,d_C_RowPtr,d_C_ColIndices);
    printf("Time elapsed for multiplication: %f seconds\n",cpuSecond()-start);

    // Return him by struct reference
    C->nnz = nnzC;
    C->csrVal = d_C;
    C->csrRowPtr = d_C_RowPtr;
    C->csrColInd =  d_C_ColIndices;


    hipFree(devCol);
    hipFree(devVal);
    hipFree(devRowPtr);

    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyMatDescr(descrC);
    hipsparseDestroy(handle);
}