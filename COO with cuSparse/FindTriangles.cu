#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * readCSV.c -- readCSV function for reading the input from
 *              a .csv file
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include <ctype.h>  
 #include <inttypes.h>
 #include <errno.h>
 #include <math.h>
 #include "readCSV.h"


__global__ void filter(cooFormat A, cooFormat C){
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = blockDim.x * gridDim.x;
    

    for (int i=index;i<C.nnz;i+=stride){
        int flag = 0;
        for (int j=0;j<A.nnz;j++){
            if ((A.cooColIndA[j] == C.cooColIndA[i]) && (A.cooRowIndA[j] == C.cooRowIndA[i])){
                flag = 1;
                break;
            }
        }
        if (!flag){
            C.cooValA[i] = 0;
        }
    }

}


__global__ void findTriangles(cooFormat A, cooFormat C, int* sum, int* counter){
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    int tid = threadIdx.x;
    int stride = blockDim.x * gridDim.x;
     
    
    // for (int s=blockDim.x/2; s>0; s>>=1) {
    //     if (tid < s) {
    //         totalSum[tid] += totalSum[tid + s];
    //     }
    //     __syncthreads();
    // }

    
    for (long i=index;i<C.nnz;i+=stride){
       for (int j=0;j<A.nnz;j++){
           if ((A.cooColIndA[j] == C.cooColIndA[i]) && (A.cooRowIndA[j] == C.cooRowIndA[i])){
               //atomicAdd(counter,1);
               atomicAdd(sum,C.cooValA[i]);
               break;
           }
       }
    //    __syncthreads();
    }
    
    __syncthreads();
    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Triangles on GPU: %d\n",sum[0]/6);
        // printf("Mphka: %d\n",*counter);
    }
    
}

__global__ void findTrianglesShared(cooFormat A, cooFormat C, int* totalSum, int* counter){

    int index = threadIdx.x + blockIdx.x*blockDim.x;
    int tid = threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int flag;

    __shared__ int rowA[1024];
    __shared__ int colA[1024];
    __shared__ int rowC[1024];
    __shared__ int colC[1024];

    if (threadIdx.x == 0 && blockIdx.x == 0){
        *totalSum = 0;
    }

    for (int i=index;i<C.nnz;i+=stride){
        
        rowA[tid] = A.cooRowIndA[index];
        colA[tid] = A.cooColIndA[index];
        rowC[tid] = C.cooRowIndA[index];
        colC[tid] = C.cooColIndA[index];

        __syncthreads();

        flag = 0;
        for (int k=0;k<1024;k++){
            if ((rowA[k] == rowC[tid]) && (colA[k] == colC[tid])){
                atomicAdd(totalSum,C.cooValA[i]);
                flag = 1;
                break;
            }
        }
        if (flag == 0){
            for (int j=0;j<A.nnz;j++){
                if ((A.cooColIndA[j] == colC[tid]) && (A.cooRowIndA[j] == rowC[tid])){
                    //atomicAdd(counter,1);
                    atomicAdd(totalSum,C.cooValA[i]);
                    break;
                }
            }
        }
        __syncthreads();
    }

    __syncthreads();

    if (threadIdx.x == 0 && blockIdx.x == 0){
        printf("Triangles on GPU with shared memory: %d\n",totalSum[0]/6);
        // printf("Mphka: %d\n",*counter);
    }

}


void findTrianglesCPU(cooFormat* A, cooFormat* C){
    int sum = 0;
    for (int i=0;i<C->nnz;i++){
       for (int j=0;j<A->nnz;j++){
           if ((A->cooColIndA[j] == C->cooColIndA[i]) && (A->cooRowIndA[j] == C->cooRowIndA[i])){
               sum += C->cooValA[i];
               break;
           }
       }
    }
    printf("Triangles on CPU: %d\n",sum/6);
}