#include "hip/hip_runtime.h"
/**********************************************************************
 *
 * readCSV.c -- readCSV function for reading the input from
 *              a .csv file
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include <ctype.h>  
 #include <inttypes.h>
 #include <errno.h>
 #include <math.h>
 #include "readCSV.h"


__global__ void filter(csrFormat A, csrFormat C, int* counter1, int* counter2){
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    if (threadIdx.x == 0 && blockIdx.x == 0){
        *counter1 = 0;
        *counter2 = 0;
    }

    for (int i=index;i<C.nnz;i+=stride){
        int flag = 0;
        for (int j=0;j<A.nnz;j++){
            if ((A.csrColInd[j] == C.csrColInd[i]) && (A.csrRowPtr[j] == C.csrRowPtr[i])){
                flag = 1;
                //atomicAdd(counter1,1);
                break;
            }
        }
        if (flag == 0){
            C.csrVal[i] = 0;
            //atomicAdd(counter2,1);
        }
    }
}


__global__ void findTriangles(csrFormat A, csrFormat C, int* sum, int N){
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = blockDim.x * gridDim.x;
     
   
    for (int i=index;i<C.nnz;i+=stride){
        for (int j=0;j<N;j++){
            for (int k=A.csrRowPtr[j];k<A.csrRowPtr[j+1];k++){
                if ((A.csrColInd[k] == C.csrColInd[i]) && (i >= C.csrRowPtr[j]) && (i < C.csrRowPtr[j+1])){
                    atomicAdd(sum,C.csrVal[i]);
                    j = N;
                    break;
                }
            }
           
        }
    }
}


__global__ void findTrianglesSum(csrFormat A, csrFormat C, int* sum, int* counter){
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int tid = threadIdx.x; 

    __shared__ int totalSum[1024];
    
    for (int i=index;i<C.nnz;i+=stride){
        totalSum[tid] = C.csrVal[index];
        __syncthreads();

        for (int s=blockDim.x/2; s>0; s>>=1) {
            if (tid < s) {
                totalSum[tid] += totalSum[tid + s];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0){
            atomicAdd(sum,totalSum[0]);
            atomicAdd(counter,1);
        }
        __syncthreads();
    }

    

    for (long i=index;i<C.nnz;i+=stride){
       
        // atomicAdd(counter,C.csrValA[i]);
        // atomicAdd(counter,1);
    }
}

void findTrianglesCPU(csrFormat* A, csrFormat* C, int N){
    int sum = 0;
    for (int i=0;i<A->nnz;i++){
        for (int j=0;j<N;j++){
            for (int k=C->csrRowPtr[j];k<C->csrRowPtr[j+1];k++){
                if ((C->csrColInd[k] == A->csrColInd[i]) && (i >= A->csrRowPtr[j]) && (i < A->csrRowPtr[j+1])){
                    sum += C->csrVal[k];
                    j = N;
                    break;
                }
            }
           
        }
    }
    printf("Triangles on CPU: %d\n",sum/6);
}