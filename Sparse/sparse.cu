#include "hip/hip_runtime.h"
/********************************************************************************
 *
 * sparse.cu -- Tester function for the csrgemm() CUDA function
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 ********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <sys/time.h>
#include "cuTrianglesFinder.h"
#include "readCSV.h"

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int main(int argc, char** argv){

    int N, M, nT_Mat;
    double matlab_time;

    /* Create the struct of type csr Format to hold the Sparse Matrices A and B */
    csrFormat h_A, d_A, 
              h_B, d_B;

    /* Create the struct of type coo Format to hold the Sparse Matrix A */
    cooFormat h_A_COO, d_A_COO;
    int* d_csrRowPtr_coo2csr;

    /* Read the input Sparse Matrix, alongside with some further info */
    readCSV(argv[1], &h_A, &h_A_COO, &N, &M, &nT_Mat, &matlab_time);

    printf("Input Data File Sample:\n");    
    printf("nnz = %d\n", h_A.nnz);
    for (int i=0;i<10;i++){
        printf("h_A.csrVal: %f\n",h_A.csrVal[i]);
        printf("h_A.csrRowPtr: %d\n",h_A.csrRowPtr[i]);
        printf("h_A.csrColInd: %d\n",h_A.csrColInd[i]);
    }

    printf("COO: Input Data File Sample:\n");    
    printf("nnz = %d\n", h_A_COO.nnz);
    for (int i=0;i<10;i++){
        printf("h_A_COO.cooVal: %f\n",h_A_COO.cooVal[i]);
        printf("h_A_COO.cooRowInd: %d\n",h_A_COO.cooRowInd[i]);
        printf("h_A_COO.cooColInd: %d\n",h_A_COO.cooColInd[i]);
    }

    /* Create the cuSPARSE handle */
    hipsparseHandle_t handle = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle));    

    /* Construct a descriptor of the matrix A */
    hipsparseMatDescr_t descrA = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    /* Construct a descriptor of the matrix A_COO */
    hipsparseMatDescr_t descrA_COO = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA_COO));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA_COO, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA_COO, HIPSPARSE_INDEX_BASE_ZERO));

    /* Construct a descriptor of the matrix B */
    hipsparseMatDescr_t descrB = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrB));
    CHECK_CUSPARSE(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO));

    /* Allocate device memory to store the sparse CSR representation of A */
    d_A.nnz = h_A.nnz;
    CUDA_CALL(hipMalloc((void **)&(d_A.csrVal),    sizeof(float) * d_A.nnz));
    CUDA_CALL(hipMalloc((void **)&(d_A.csrRowPtr), sizeof(int) * (N + 1)));
    CUDA_CALL(hipMalloc((void **)&(d_A.csrColInd), sizeof(int) * d_A.nnz));

    /* Copy the sparse CSR representation of A from the Host to the Device */
    CUDA_CALL(hipMemcpy(d_A.csrVal,    h_A.csrVal,    d_A.nnz * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_A.csrRowPtr, h_A.csrRowPtr, (N + 1) * sizeof(int)  , hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_A.csrColInd, h_A.csrColInd, d_A.nnz * sizeof(int)  , hipMemcpyHostToDevice));

    /* Allocate device memory to store the sparse COO representation of A */
    d_A_COO.nnz = h_A_COO.nnz;
    CUDA_CALL(hipMalloc((void **)&(d_A_COO.cooVal),    sizeof(float) * d_A_COO.nnz));
    CUDA_CALL(hipMalloc((void **)&(d_A_COO.cooRowInd), sizeof(int) * d_A_COO.nnz));
    CUDA_CALL(hipMalloc((void **)&(d_csrRowPtr_coo2csr), sizeof(int) * (N + 1)));
    CUDA_CALL(hipMalloc((void **)&(d_A_COO.cooColInd), sizeof(int) * d_A_COO.nnz));

    /* Copy the sparse COO representation of A from the Host to the Device */
    CUDA_CALL(hipMemcpy(d_A_COO.cooVal,    h_A_COO.cooVal,    d_A_COO.nnz * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_A_COO.cooRowInd, h_A_COO.cooRowInd, d_A_COO.nnz * sizeof(int)  , hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_A_COO.cooColInd, h_A_COO.cooColInd, d_A_COO.nnz * sizeof(int)  , hipMemcpyHostToDevice));

    CHECK_CUSPARSE(hipsparseXcoo2csr(handle,
                    d_A_COO.cooRowInd,
                    d_A_COO.nnz,
                    N,
                    d_csrRowPtr_coo2csr,
                    HIPSPARSE_INDEX_BASE_ZERO ));

    CUDA_CALL(hipMemcpy(h_A_COO.cooRowInd, d_csrRowPtr_coo2csr, (N + 1) * sizeof(int), hipMemcpyDeviceToHost));

    printf("Input Data File Sample:\n");    
    printf("nnz = %d\n", h_A.nnz);
    for (int i=0;i<10;i++){
        printf("h_A.csrVal: %f\n",h_A.csrVal[i]);
        printf("h_A.csrRowPtr: %d\n",h_A.csrRowPtr[i]);
        printf("h_A.csrColInd: %d\n",h_A.csrColInd[i]);
    }

    printf("COO2CSR: Input Data File Sample:\n");    
    printf("nnz = %d\n", h_A_COO.nnz);
    for (int i=0;i<10;i++){
        printf("h_A_COO.cooVal: %f\n",h_A_COO.cooVal[i]);
        printf("h_A_COO.cooRowInd: %d\n",h_A_COO.cooRowInd[i]);
        printf("h_A_COO.cooColInd: %d\n",h_A_COO.cooColInd[i]);
    }

    int i;
    for(i=0;i<h_A.nnz;i++)
    {
        if ( (h_A.csrVal[i] != h_A_COO.cooVal[i]) || (h_A.csrColInd[i] != h_A_COO.cooColInd[i]) )
            printf("Col ERROR\n");

        if (i < N + 1)
            if ( h_A.csrRowPtr[i] != h_A_COO.cooRowInd[i])
                printf("Row ERROR\n");
    }
    printf("h_A.nnz = %d = %d = i\n", h_A.nnz, i);

    /* Cleanup */
    CUDA_CALL(hipFree(d_csrRowPtr_coo2csr));
    CUDA_CALL(hipFree(d_A_COO.cooVal));    CUDA_CALL(hipFree(d_A_COO.cooRowInd));     CUDA_CALL(hipFree(d_A_COO.cooColInd));
    free(h_A_COO.cooVal);                   free(h_A_COO.cooRowInd);                    free(h_A_COO.cooColInd);

    int baseB;
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &(d_B.nnz);
    CHECK_CUSPARSE(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

    // Allocate device memory to store the Row Pointers of the sparse CSR representation of B
    CUDA_CALL(hipMalloc((void**)&(d_B.csrRowPtr), sizeof(int)*(N+1)));

    /* First determine the nnz of Sparse Matrix B */
    CHECK_CUSPARSE(hipsparseXcsrgemmNnz(handle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        N, 
                        N, 
                        N,
                        descrA, 
                        d_A.nnz,
                        d_A.csrRowPtr, 
                        d_A.csrColInd,
                        descrA, 
                        d_A.nnz,
                        d_A.csrRowPtr, 
                        d_A.csrColInd,
                        descrB, 
                        d_B.csrRowPtr,
                        nnzTotalDevHostPtr ));

    if (NULL != nnzTotalDevHostPtr){
        d_B.nnz = *nnzTotalDevHostPtr;
    } else {
        CUDA_CALL(hipMemcpy(&(d_B.nnz), d_B.csrRowPtr+N, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(&baseB,     d_B.csrRowPtr,   sizeof(int), hipMemcpyDeviceToHost));
        d_B.nnz -= baseB;
    }

    /* Allocate device memory to store the rest of the sparse CSR representation of B */
    CUDA_CALL(hipMalloc((void**)&d_B.csrVal,    sizeof(float) * d_B.nnz));
    CUDA_CALL(hipMalloc((void**)&d_B.csrColInd, sizeof(int) * d_B.nnz));

                        /* Timer variable */
                        double first = cpuSecond();

    /* Perform the actual multiplication A * A = B */
    CHECK_CUSPARSE(hipsparseScsrgemm(handle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        N, 
                        N, 
                        N,
                        descrA, 
                        d_A.nnz,
                        d_A.csrVal,
                        d_A.csrRowPtr, 
                        d_A.csrColInd,
                        descrA, 
                        d_A.nnz,
                        d_A.csrVal,
                        d_A.csrRowPtr, 
                        d_A.csrColInd,
                        descrB,
                        d_B.csrVal,
                        d_B.csrRowPtr, 
                        d_B.csrColInd ));

                        /* Timer display */
                        printf("GPU Sparse Matrices Multiplication wall clock time: %fs\n",cpuSecond()-first);


    /* Allocate memory onto the Host to hold the sparse CSR representation of B */
    h_B.nnz = d_B.nnz;
    h_B.csrVal = (float*)malloc ((h_B.nnz) * sizeof(float));
    h_B.csrRowPtr = (int*)malloc ((N + 1) * sizeof(int));
    h_B.csrColInd = (int*)malloc ((h_B.nnz) * sizeof(int));

    /* Copy the sparse CSR representation of B from the Device back to the Host */
    CUDA_CALL(hipMemcpy(h_B.csrVal,    d_B.csrVal,    d_B.nnz * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_B.csrRowPtr, d_B.csrRowPtr, (N + 1) * sizeof(int)  , hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(h_B.csrColInd, d_B.csrColInd, d_B.nnz * sizeof(int)  , hipMemcpyDeviceToHost));


    printf("Input Data File Sample:\n");    
    printf("h_B.nnz = %d\n", d_B.nnz);
    for (int i=0;i<10;i++){
        printf("h_B.csrVal: %f\n",     h_B.csrVal[i]);
        printf("h_B.csrRowPtr: %d\n",  h_B.csrRowPtr[i]);
        printf("h_B.csrColInd: %d\n",  h_B.csrColInd[i]);
    }

    /* Cleanup */
    CUDA_CALL(hipFree(d_A.csrVal));          CUDA_CALL(hipFree(d_B.csrVal));
    CUDA_CALL(hipFree(d_A.csrRowPtr));       CUDA_CALL(hipFree(d_B.csrRowPtr));
    CUDA_CALL(hipFree(d_A.csrColInd));       CUDA_CALL(hipFree(d_B.csrColInd));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}
