#include "hip/hip_runtime.h"
/********************************************************************************
 *
 * sparse.cu -- Tester function for the csrgemm() CUDA function
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 ********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <sys/time.h>
#include "cuTrianglesFinder.h"
#include "readCSV.h"

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void pri(float* tmp){
    printf("Random value: %f\n",tmp[4]);
}

int main(int argc, char** argv){

    int N, M, nT_Mat;
    double matlab_time;

    /* Create the struct of type csr Format to hold the Sparse Matrices A and B */
    csrFormat h_A, d_A, 
              h_B, d_B;

    /* Read the input Sparse Matrix, alongside with some further info */
    read_cSV(argv[1], &h_A, &N, &M, &nT_Mat, &matlab_time);

    /* Create the cuSPARSE handle */
    hipsparseHandle_t handle = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle));    

    /* Construct a descriptor of the matrix A */
    hipsparseMatDescr_t descrA = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    /* Construct a descriptor of the matrix B */
    hipsparseMatDescr_t descrB = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrB));
    CHECK_CUSPARSE(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO));

    /* Allocate device memory to store the sparse CSR representation of A */
    CUDA_CALL(hipMalloc((void **)&(d_A.csrVal),    sizeof(float) * h_A.nnz));
    CUDA_CALL(hipMalloc((void **)&(d_A.csrRowPtr), sizeof(int) * (N + 1)));
    CUDA_CALL(hipMalloc((void **)&(d_A.csrColInd), sizeof(int) * h_A.nnz));

    int baseB;
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &(h_B.nnz);
    CHECK_CUSPARSE(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

    // Allocate device memory to store the Row Pointers of the sparse CSR representation of B
    CUDA_CALL(hipMalloc((void**)&(d_B.csrRowPtr), sizeof(int)*(N+1)));

    /* First determine the nnz of Sparse Matrix B */
    CHECK_CUSPARSE(hipsparseXcsrgemmNnz(hipsparseHandle_t handle,
                        hipsparseOperation_t HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        hipsparseOperation_t HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        int N, 
                        int N, 
                        int N,
                        const hipsparseMatDescr_t descrA, 
                        const int h_A.nnz,
                        const int d_A.csrRowPtr, 
                        const int d_A.csrColInd,
                        const hipsparseMatDescr_t descrA, 
                        const int h_A.nnz,
                        const int d_A.csrRowPtr, 
                        const int d_A.csrColInd,
                        const hipsparseMatDescr_t descrB, 
                        int d_B.csrRowPtr,
                        int nnzTotalDevHostPtr ));

    if (NULL != nnzTotalDevHostPtr){
        d_B.nnz = *nnzTotalDevHostPtr;
    } else {
        CUDA_CALL(hipMemcpy(&(d_B.nnz), d_B.csrRowPtr+N, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(&baseB,     d_B.csrRowPtr, sizeof(int), hipMemcpyDeviceToHost));
        d_B.nnz -= baseB;
    }

    /* Allocate device memory to store the rest of the sparse CSR representation of B */
    CUDA_CALL(hipMalloc((void**)&d_B.csrColInd, sizeof(int) * d_B.nnz));
    CUDA_CALL(hipMalloc((void**)&d_B.csrVal,    sizeof(float) * d_B.nnz));

    /* Perform the actual multiplication A * A = B */
    CHECK_CUSPARSE(hipsparseScsrgemm(hipsparseHandle_t handle,
                        hipsparseOperation_t HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        hipsparseOperation_t HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        int N, 
                        int N, 
                        int N,
                        const hipsparseMatDescr_t descrA, 
                        const int h_A.nnz,
                        const float d_A.csrVal,
                        const int d_A.csrRowPtr, 
                        const int d_A.csrColInd,
                        const hipsparseMatDescr_t descrA, 
                        const int h_A.nnz,
                        const float d_A.csrVal,
                        const int d_A.csrRowPtr, 
                        const int d_A.csrColInd,
                        const hipsparseMatDescr_t descrB,
                        float d_B.csrVal,
                        const int d_B.csrRowPtr, 
                        int d_B.csrColInd ));

    /* Maybe transfer B matrix to the host and print some of the results */


    /* Cleanup */

    /***** Add any other cleanup here *****/

    CUDA_CALL(hipFree(d_csrValA));          CUDA_CALL(hipFree(d_csrValB));
    CUDA_CALL(hipFree(d_csrRowPtrA));       CUDA_CALL(hipFree(d_csrRowPtrB));
    CUDA_CALL(hipFree(d_csrColIndA));       CUDA_CALL(hipFree(d_csrColIndB));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}
