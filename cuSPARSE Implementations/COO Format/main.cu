/**********************************************************************
 *
 * readCSV.c -- readCSV function for reading the input from
 *              a .csv file
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include <ctype.h>  
 #include <inttypes.h>
 #include <errno.h>
 #include <math.h>
 #include "readCSV.h"
 #include "validation.h"
 

int main(int argc, char** argv){
 
    char* fName = argv[1];
    int N, M, nT_Mat;
    double matlab_time;

    cooFormat A, C;

    printf("Reading of dataset and validation file has started\n");
    readCSV(fName, &A, &N, &M, &nT_Mat, &matlab_time);
    printf("Reading of dataset and validation file has ended\n");

    printf("Nonzeros = %d\n", A.nnz);
    
    printf("Validation File:\n N = %d, M = %d\n Matlab result was %d, produced in %lf\n", N, M, nT_Mat, matlab_time);

    // Multiplication of two sparse arrays using cuSparse
    mulSparse(&A,&C,N);

    // Move adjancy array to gpu memory
    float* devVal;
    int* devCol, *devRow;
    int nnzA = A.nnz;
    CHECK(hipMalloc((void**)&devVal,nnzA*sizeof(float)));
    CHECK(hipMalloc((void**)&devCol,nnzA*sizeof(int)));
    CHECK(hipMalloc((void**)&devRow,nnzA*sizeof(int)));
    CHECK(hipMemcpy(devVal,A.cooValA,nnzA*sizeof(float),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(devCol,A.cooColIndA,nnzA*sizeof(int),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(devRow,A.cooRowIndA,nnzA*sizeof(int),hipMemcpyHostToDevice));

    // Use a struct to access it easier
    cooFormat B;
    B.cooColIndA = devCol;
    B.cooRowIndA = devRow;
    B.cooValA = devVal;
    B.nnz = A.nnz;

    // Initialize value of triangles to 0
    int* sum, *h_sum;
    CHECK(hipMalloc((void**)&sum,sizeof(int)));
    h_sum = (int*)malloc(sizeof(int));
    *h_sum = 0;
    CHECK(hipMemcpy(sum,h_sum,sizeof(int),hipMemcpyHostToDevice));
    
    // double st1 = cpuSecond();
    // filter<<<160,1024>>>(B,C,sum,counter2);
    // CHECK(hipPeekAtLastError());
    // CHECK(hipDeviceSynchronize());
    // printf("Time filtering on GPU: %lf sec\n",cpuSecond()-st1);
    // printf("Vrhka tosa simeia: %d kai midenisa tosa: %d\n",*counter1,*counter2);

    // double st2 = cpuSecond();
    // findTrianglesShared<<<160,1024>>>(B,C,sum,counter);
    // CHECK(hipPeekAtLastError());
    // CHECK(hipDeviceSynchronize());
    // printf("Time on GPU using shared memory: %lf sec\n",cpuSecond()-st2);

    
    // Call kernel to calculate number of triangles
    double st3 = cpuSecond();
    findTriangles<<<20,1024>>>(B,C,sum);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    hipMemcpy(h_sum,sum,sizeof(int),hipMemcpyDeviceToHost);
    printf("Triangles using COO format: %d\n",h_sum[0]/3);
    printf("Time on GPU using COO format: %lf sec\n",cpuSecond()-st3);
    
    if (validation(h_sum[0]/3,nT_Mat)){
        printf("Validation on GPU: PASSED\n");
    }

    // Move C to host memory to call cpu function
    // float* h_val = (float*)malloc(C.nnz*sizeof(float));
    // int* h_col = (int*)malloc(C.nnz*sizeof(int));
    // int* h_row = (int*)malloc(C.nnz*sizeof(int));
    // CHECK(hipMemcpy(h_val,C.cooValA,C.nnz*sizeof(float),hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(h_col,C.cooColIndA,C.nnz*sizeof(int),hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(h_row,C.cooRowIndA,C.nnz*sizeof(int),hipMemcpyDeviceToHost));

    //// Pass it through a struct
    // cooFormat D;
    // D.cooColIndA = h_col;
    // D.cooRowIndA = h_row;
    // D.cooValA = h_val;
    // D.nnz = C.nnz;


    // double st = cpuSecond();
    // int triangles = findTrianglesCPU(&A,&D);
    // printf("Time on CPU: %lf sec\n",cpuSecond()-st);

    // if (validation(triangles,nT_Mat)){
    //     printf("Validation on CPU: PASSED\n");
    // }
    
    
    free(h_sum);
    free(h_col);
    free(h_val);
    free(h_row);
    hipFree(devCol);
    hipFree(devRow);
    hipFree(devVal);
    hipFree(C.cooValA);
    hipFree(C.cooColIndA);
    hipFree(C.cooRowIndA);
    hipFree(sum);

    return 0; 
}