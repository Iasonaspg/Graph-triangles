#include "hip/hip_runtime.h"
/********************************************************************************
 *
 * sparse.cu -- Tester function for the csrgemm() CUDA function
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 ********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <sys/time.h>
#include "readCSV.h"
#include "cuFindTriangles.h"
#include "validation.h"


int main(int argc, char** argv){

    int *h_nT, nT_Mat, N, M,
        *d_nT;

    double matlab_time;

    char fullValidationFlag = 0;

    /* Create the struct of type csr Format to hold the Sparse Matrices A and B */
    csrFormat h_A, d_A, d_B;

    /* Parsing input arguments */
    if ( argc < 4 ) {
        printf("--Reading Input Data from CSV file: Started--\n");    
        readCSV(argv[1], &h_A, &N, &M, &nT_Mat, &matlab_time);
        printf("--Reading Input Data from CSV file: DONE!--\n");   
        if ( argc == 3 )
            if ( strcmp(argv[2], "--fullVal") == 0 )              
                fullValidationFlag = 1; 
        // -------- Do not use when timing --------
    } else {
        printf("Usage: ./trianglesGPU <CSVfileName> <--fullVal>\n");
        printf(" where <CSVfileName>.csv is the name of the input data file (auto | great-britain_osm | delaunay_n22 | delaunay_n10)\n");
        printf("No need for suffix '.csv'\n");
        printf("where <--fullVal> is the verbose flag for validation at quite every stage of the program\n");
        exit(1);
    }


    /* CUDA Device setup */
    size_t threadsPerBlock, warp;
    size_t numberOfBlocks, SMs;
    hipError_t err;
    int deviceId;
    hipDeviceProp_t props;
    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&props, deviceId);
    warp = props.warpSize;
    SMs = props.multiProcessorCount;
    // 1D threads & blocks
    threadsPerBlock = 8 * warp;
    numberOfBlocks  = 5 * SMs;


    /* Create the cuSPARSE handle */
    hipsparseHandle_t handle = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle));    

    /* Construct a descriptor of the matrix A */
    hipsparseMatDescr_t descrA = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA));
    CHECK_CUSPARSE(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

    /* Construct a descriptor of the matrix B */
    hipsparseMatDescr_t descrB = 0;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrB));
    CHECK_CUSPARSE(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO));

    /* Allocate device memory to store the sparse CSR representation of A */
    d_A.nnz = h_A.nnz;
    CUDA_CALL(hipMalloc((void **)&(d_A.csrVal),    sizeof(float) * d_A.nnz));
    CUDA_CALL(hipMalloc((void **)&(d_A.csrRowPtr), sizeof(int) * (N + 1)));
    CUDA_CALL(hipMalloc((void **)&(d_A.csrColInd), sizeof(int) * d_A.nnz));

    /* Copy the sparse CSR representation of A from the Host to the Device */
    CUDA_CALL(hipMemcpy(d_A.csrVal,    h_A.csrVal,    d_A.nnz * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_A.csrRowPtr, h_A.csrRowPtr, (N + 1) * sizeof(int)  , hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(d_A.csrColInd, h_A.csrColInd, d_A.nnz * sizeof(int)  , hipMemcpyHostToDevice));
    
    int baseB;
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &(d_B.nnz);
    CHECK_CUSPARSE(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

    // Allocate device memory to store the Row Pointers of the sparse CSR representation of B
    CUDA_CALL(hipMalloc((void**)&(d_B.csrRowPtr), sizeof(int)*(N+1)));


    /* Timer variables setup */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

            /* Begin timer */
            hipEventRecord(start);

    /* First determine the nnz of Sparse Matrix B */
    CHECK_CUSPARSE(hipsparseXcsrgemmNnz(handle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        N, 
                        N, 
                        N,
                        descrA, 
                        d_A.nnz,
                        d_A.csrRowPtr, 
                        d_A.csrColInd,
                        descrA, 
                        d_A.nnz,
                        d_A.csrRowPtr, 
                        d_A.csrColInd,
                        descrB, 
                        d_B.csrRowPtr,
                        nnzTotalDevHostPtr ));

    if (NULL != nnzTotalDevHostPtr){
        d_B.nnz = *nnzTotalDevHostPtr;
    } else {
        CUDA_CALL(hipMemcpy(&(d_B.nnz), d_B.csrRowPtr+N, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(&baseB,     d_B.csrRowPtr,   sizeof(int), hipMemcpyDeviceToHost));
        d_B.nnz -= baseB;
    }

    /* Allocate device memory to store the rest of the sparse CSR representation of B */
    CUDA_CALL(hipMalloc((void**)&d_B.csrVal,    sizeof(float) * d_B.nnz));
    CUDA_CALL(hipMalloc((void**)&d_B.csrColInd, sizeof(int) * d_B.nnz));

    /* Perform the actual multiplication A * A = B */
    CHECK_CUSPARSE(hipsparseScsrgemm(handle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        N, 
                        N, 
                        N,
                        descrA, 
                        d_A.nnz,
                        d_A.csrVal,
                        d_A.csrRowPtr, 
                        d_A.csrColInd,
                        descrA, 
                        d_A.nnz,
                        d_A.csrVal,
                        d_A.csrRowPtr, 
                        d_A.csrColInd,
                        descrB,
                        d_B.csrVal,
                        d_B.csrRowPtr, 
                        d_B.csrColInd ));

    /* Allocating memory to hold the nT variable (number of Triangles) */
    CUDA_CALL(hipMalloc(&d_nT, 1 * sizeof(int)));
    h_nT = (int*)malloc (1 * sizeof(int));

    /* Zero out the content of the variable, 
    so that the summation result is valid */
    cuZeroVariable<<<1,1>>>( d_nT );

    CUDA_CALL(hipDeviceSynchronize());

    /* Hadamard Product Manually */
    // Calculating the Number of Triangles (nT) through the kernel 
    // (Only the sumation is performed here, the *(1/6) will be executed later on)
    cuTrianglesFinderHadamardOnly<<<numberOfBlocks, threadsPerBlock>>>
    (d_A, d_B, N, d_nT);

            err = hipGetLastError();
            if (err != hipSuccess) {
                printf("Error \"%s\" at %s:%d\n", hipGetErrorString(err),
                       __FILE__,__LINE__);
                return EXIT_FAILURE;
            }

    CUDA_CALL(hipDeviceSynchronize());

            /* Stop Timer */
            hipEventRecord(stop);
            hipEventSynchronize(stop);

    /* Copying nT, as calculated on GPU, back to the CPU */
    CUDA_CALL(hipMemcpy(h_nT, d_nT, 1 * sizeof(int), hipMemcpyDeviceToHost));

    /* Validating the result */
    // Executing the nT = nT*(1/6), that was omitted in cuFindTriangles
    int pass = validation(*h_nT/3, nT_Mat);    
    // though as of condition if ( col>row ) we have cut additions to half
    // due to the symmetry of the adjacency matrix, so 2*(nT/6) = nT/3
    assert(pass != 0);

    /* Calculate elapsed time */
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    /* Timer display */
    printf("  -GPU number of triangles nT: %d, Wall clock time: %fms ( < %lf ( Matlab Time ) )\n", *h_nT/3, milliseconds, matlab_time);

            /* Write the results into file */
            FILE *fp;
            fp = fopen("GPU_Results.txt", "a");
            if ( fp == NULL ) {
              perror("Failed: Opening file Failed\n");
              return 1;
            }
            fprintf(fp, "%f\n", milliseconds);
            fclose(fp);
    
    if ( fullValidationFlag ) {

        /* Create the struct of type csr Format to hold the Sparse Matrix B */
        csrFormat h_B;
        /* Create the struct of type coo Format to hold the Sparse Matrix A */
        cooFormat h_A_COO, d_A_COO, 
                  h_B_COO, d_B_COO,
                  h_B_COO_Mat;

        /* Define the nnz of the COO, same as in CSR */
        h_A_COO.nnz = h_A.nnz;

        /* Allocate memory onto the Host to hold the sparse CSR representation of B */
        h_B.nnz = d_B.nnz;
        h_B.csrVal = (float*)malloc ((h_B.nnz) * sizeof(float));
        h_B.csrRowPtr = (int*)malloc ((N + 1) * sizeof(int));
        h_B.csrColInd = (int*)malloc ((h_B.nnz) * sizeof(int));

        h_B_COO_Mat.nnz = h_B.nnz;

        /* Read A and B in COO format as found through Matlab from .csv */
        readCSV_COO(argv[1], &h_A_COO, &h_B_COO_Mat);

        int* d_csrRowPtr_coo2csr;

        /* Construct a descriptor of the matrix A_COO */
        hipsparseMatDescr_t descrA_COO = 0;
        CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrA_COO));
        CHECK_CUSPARSE(hipsparseSetMatType(descrA_COO, HIPSPARSE_MATRIX_TYPE_GENERAL));
        CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrA_COO, HIPSPARSE_INDEX_BASE_ZERO));
        /* Construct a descriptor of the matrix A_COO */
        hipsparseMatDescr_t descrB_COO = 0;
        CHECK_CUSPARSE(hipsparseCreateMatDescr(&descrB_COO));
        CHECK_CUSPARSE(hipsparseSetMatType(descrB_COO, HIPSPARSE_MATRIX_TYPE_GENERAL));
        CHECK_CUSPARSE(hipsparseSetMatIndexBase(descrB_COO, HIPSPARSE_INDEX_BASE_ZERO));

        /* Allocate device memory to store the sparse COO representation of A */
        d_A_COO.nnz = h_A_COO.nnz;
        CUDA_CALL(hipMalloc((void **)&(d_A_COO.cooVal),    sizeof(float) * d_A_COO.nnz));
        CUDA_CALL(hipMalloc((void **)&(d_A_COO.cooRowInd), sizeof(int) * d_A_COO.nnz));
        CUDA_CALL(hipMalloc((void **)&(d_csrRowPtr_coo2csr), sizeof(int) * (N + 1)));
        CUDA_CALL(hipMalloc((void **)&(d_A_COO.cooColInd), sizeof(int) * d_A_COO.nnz));


        /* Copy the sparse COO representation of A from the Host to the Device */
        CUDA_CALL(hipMemcpy(d_A_COO.cooVal,    h_A_COO.cooVal,    d_A_COO.nnz * sizeof(float), hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(d_A_COO.cooRowInd, h_A_COO.cooRowInd, d_A_COO.nnz * sizeof(int)  , hipMemcpyHostToDevice));
        CUDA_CALL(hipMemcpy(d_A_COO.cooColInd, h_A_COO.cooColInd, d_A_COO.nnz * sizeof(int)  , hipMemcpyHostToDevice));

        CHECK_CUSPARSE(hipsparseXcoo2csr(handle,
                        d_A_COO.cooRowInd,
                        d_A_COO.nnz,
                        N,
                        d_csrRowPtr_coo2csr,
                        HIPSPARSE_INDEX_BASE_ZERO ));

        CUDA_CALL(hipMemcpy(h_A_COO.cooRowInd, d_csrRowPtr_coo2csr, (N + 1) * sizeof(int), hipMemcpyDeviceToHost));

        int i;
        for(i=0;i<h_A.nnz;i++)
        {
            if ( (h_A.csrVal[i] != h_A_COO.cooVal[i]) || (h_A.csrColInd[i] != h_A_COO.cooColInd[i]) )
                printf("Col ERROR\n");

            if ( i < N + 1 )
                if ( h_A.csrRowPtr[i] != h_A_COO.cooRowInd[i])
                    printf("Row ERROR\n");
        }
        printf("h_A.nnz = %d = %d = i\n", h_A.nnz, i);

        /* Cleanup */
        CUDA_CALL(hipFree(d_csrRowPtr_coo2csr));
        CUDA_CALL(hipFree(d_A_COO.cooVal));    CUDA_CALL(hipFree(d_A_COO.cooRowInd));     CUDA_CALL(hipFree(d_A_COO.cooColInd));
        free(h_A_COO.cooVal);                   free(h_A_COO.cooRowInd);                    free(h_A_COO.cooColInd);

        /* Allocate device memory to store the sparse COO representation of B */
        h_B_COO.nnz = h_B.nnz;
        d_B_COO.nnz = h_B_COO.nnz;
        // CUDA_CALL(hipMalloc((void **)&(d_B_COO.cooVal),    sizeof(float) * d_B_COO.nnz));
        CUDA_CALL(hipMalloc((void **)&(d_B_COO.cooRowInd), sizeof(int) * d_B_COO.nnz));
        // CUDA_CALL(hipMalloc((void **)&(d_B_COO.cooColInd), sizeof(int) * d_B_COO.nnz));

        /* Allocating memory onto the Host to hold the struct of Sparse Matrix B */
        h_B_COO.cooVal = (float*)malloc ((h_B_COO.nnz)*sizeof(float));
        h_B_COO.cooRowInd = (int*)malloc ((h_B_COO.nnz)*sizeof(int));
        h_B_COO.cooColInd = (int*)malloc ((h_B_COO.nnz)*sizeof(int));

        /* Copy the sparse CSR representation of B from the Device back to the Host */
        CUDA_CALL(hipMemcpy(h_B.csrVal,    d_B.csrVal,    d_B.nnz * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(h_B.csrRowPtr, d_B.csrRowPtr, (N + 1) * sizeof(int)  , hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(h_B.csrColInd, d_B.csrColInd, d_B.nnz * sizeof(int)  , hipMemcpyDeviceToHost));

        printf("Input Data File Sample:\n");    
        printf("h_B.nnz = %d = %d\n", d_B.nnz, h_B.csrRowPtr[N]);
        for (int i=0;i<10;i++){
            printf("h_B.csrVal: %f\n",     h_B.csrVal[i]);
            printf("h_B.csrRowPtr: %d\n",  h_B.csrRowPtr[i]);
            printf("h_B.csrColInd: %d\n",  h_B.csrColInd[i]);
        }

        CHECK_CUSPARSE(hipsparseXcsr2coo(handle, 
                                        d_B.csrRowPtr,
                                        d_B.nnz, N, 
                                        d_B_COO.cooRowInd,
                                        HIPSPARSE_INDEX_BASE_ZERO ));

        /* Copy the sparse CSR representation of B from the Device back to the Host */
        CUDA_CALL(hipMemcpy(h_B_COO.cooVal,    d_B.csrVal,         d_B_COO.nnz * sizeof(float), hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(h_B_COO.cooRowInd, d_B_COO.cooRowInd,  d_B_COO.nnz * sizeof(int)  , hipMemcpyDeviceToHost));
        CUDA_CALL(hipMemcpy(h_B_COO.cooColInd, d_B.csrColInd,      d_B_COO.nnz * sizeof(int)  , hipMemcpyDeviceToHost));


        for(i=0;i<h_B_COO.nnz;i++)
        {
            if ( h_B_COO_Mat.cooVal[i] != h_B_COO.cooVal[i] )
                printf("Val ERROR\n");

            if ( h_B_COO_Mat.cooRowInd[i] != h_B_COO.cooRowInd[i])
                printf("Row ERROR\n");

            if (h_B_COO_Mat.cooColInd[i] != h_B_COO.cooColInd[i])
                printf("Col ERROR\n");
        }
        printf("h_B.nnz = %d = %d = i = %d = d_B_COO.nnz = %d = h_B_COO_Mat.nnz\n", h_B.nnz, i, d_B_COO.nnz, h_B_COO_Mat.nnz);
    }

    /* Cleanup */
    CUDA_CALL(hipFree(d_A.csrVal));          CUDA_CALL(hipFree(d_B.csrVal));
    CUDA_CALL(hipFree(d_A.csrRowPtr));       CUDA_CALL(hipFree(d_B.csrRowPtr));
    CUDA_CALL(hipFree(d_A.csrColInd));       CUDA_CALL(hipFree(d_B.csrColInd));

    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrA));
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descrB));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0;
}
