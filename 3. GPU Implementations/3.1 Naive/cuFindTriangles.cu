#include "hip/hip_runtime.h"
/******************************************************************************
 *
 * cuFindTriangles.cu -- The kernel that calculates the Number of triangles into
 *                       a graph given the CSR format of its Adjacency Matrix
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 ******************************************************************************/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "readCSV.h"
#include "cuFindTriangles.h"

__global__
/* Kernel function that zeros the number of triangles variable */
void cuZeroVariable(int* nT) {

  (*nT) = 0;
}

__global__
/* Kernel function that finds the number of triangles formed in the graph */
void cuFindTriangles(csrFormat A, int N, int* nT) {

  // Each thread processes a different row
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  int stride = blockDim.x * gridDim.x;

  // Iterate over rows
  for (int row = index; row < N; row += stride) {

      // Iterate over columns
      for (int j = A.csrRowPtr[row]; j < A.csrRowPtr[row+1]; j++) {

        int col = A.csrColInd[j];
        // [row, col] = position of 1 horizontally

        if ( col>row ) {
        // OPTIMIZATION: Due to symmetry, nT of the upper half array is
        // equal to half the nT, thus additions are cut down to half !           
          int beginPtr_csr_row = A.csrRowPtr[row];
          int beginPtr_csc_col = A.csrRowPtr[col];
          // Multiplication of A[:,col] * A[row,:]      
          for (int k = beginPtr_csc_col; k < A.csrRowPtr[col+1]; k++) {
                  
            int csc_row = A.csrColInd[k];
            // [csr_row, k] = position of 1 vertically

            for (int l = beginPtr_csr_row; l < A.csrRowPtr[row+1]; l++) {
  
                int csr_col = A.csrColInd[l];

                if ( csc_row == csr_col )
                    atomicAdd( nT, 1 );
                else if ( csr_col > csc_row ) {
                    // OPTIMIZATION: when col>row no need to go further,
                    // continue to the next col, plus for further optimization
                    // keep track of the beginPtr_csr_row where the previous
                    // iteration stopped, so that no time is wasted in rechecking
                    beginPtr_csr_row = l;
                    break;
                }

            }
          }
        }
      }
  }

}


/*
// **************  2D threads & blocks  **************
{

  int val = 0;

  int row_index = blockIdx.x * blockDim.x + threadIdx.x;
  int col_index = blockIdx.y * blockDim.y + threadIdx.y;
  int row_stride = blockDim.x * gridDim.x;
  int col_stride = blockDim.y * gridDim.y;

  for(int row = row_index; row < N; row += row_stride)
  {
    for(int col = col_index; col < N; col += col_stride)
      if ( (A[row * N + col] != 0) )
      {
        val = 0;
        for ( int k = 0; k < N; ++k )
          val += A[row * N + k] * A[k * N + col];
        B[row * N + col] = val;
      }
  }
}
// ***************************************************
*/
