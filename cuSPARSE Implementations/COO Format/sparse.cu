#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <sys/time.h>
#include "readCSV.h"


double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void mulSparse(cooFormat* A, cooFormat* C, int N){

    // Initialize cuSPARSE
    hipsparseHandle_t handle;   
    hipsparseCreate(&handle);

    int nnzA = A->nnz;

    // Bring array to gpu memory
    float* devVal;
    int* devCol, *devRow;
    hipMalloc((void**)&devVal,nnzA*sizeof(float));
    hipMalloc((void**)&devCol,nnzA*sizeof(int));
    hipMalloc((void**)&devRow,nnzA*sizeof(int));
    hipMemcpy(devVal,A->cooValA,nnzA*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(devCol,A->cooColIndA,nnzA*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devRow,A->cooRowIndA,nnzA*sizeof(int),hipMemcpyHostToDevice);

    // Convert to CSR format for cuSparse needs
    int* csrRowPtrA;
    hipMallocManaged(&csrRowPtrA,(N+1)*sizeof(int));
    hipsparseXcoo2csr(handle,devRow,nnzA,N,csrRowPtrA,HIPSPARSE_INDEX_BASE_ZERO);
    

    // Descriptor for sparse matrix A and C
    hipsparseMatDescr_t descrA,descrC;     
    hipsparseCreateMatDescr(&descrA);
    hipsparseCreateMatDescr(&descrC);

    int* d_C_RowPtr;
    hipMalloc((void**)&d_C_RowPtr,(N+1)*sizeof(*d_C_RowPtr));

    // Calculate the number of C's non-zero values
    int nnzC;
    //hipMallocManaged(&nnzC,sizeof(int));
    hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseXcsrgemmNnz(handle,transA,transA,N,N,N,descrA,nnzA,csrRowPtrA,devCol,descrA,nnzA,csrRowPtrA,devCol,descrC,d_C_RowPtr,&nnzC);

    printf("Non zero of C: %d\n",nnzC);

    // Allocate memory for array C
    float* d_C;
    int *d_C_ColIndices;
    hipMalloc((void**)&d_C, (nnzC)*sizeof(*d_C));
    hipMalloc((void**)&d_C_ColIndices, (nnzC) * sizeof(*d_C_ColIndices));
    
    // Calculate the multiplication A*A
    double start = cpuSecond();
    hipsparseScsrgemm(handle,transA,transA,N,N,N,descrA,nnzA,devVal,csrRowPtrA,devCol,descrA,nnzA,devVal,csrRowPtrA,devCol,descrC,d_C,d_C_RowPtr,d_C_ColIndices);
    printf("Time elapsed for multiplication: %f seconds\n",cpuSecond()-start);

    // Convert row array from CSR to COO format
    int* cooRowC;
    hipMalloc((void**)&cooRowC,(nnzC)*sizeof(int));
    hipsparseXcsr2coo(handle,d_C_RowPtr,(nnzC),N,cooRowC,HIPSPARSE_INDEX_BASE_ZERO);

    C->nnz = nnzC;
    C->cooValA = d_C;
    C->cooRowIndA = cooRowC;
    C->cooColIndA =  d_C_ColIndices;

    hipFree(devCol);
    hipFree(devVal);
    hipFree(devRow);

    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyMatDescr(descrC);
    hipsparseDestroy(handle);
}