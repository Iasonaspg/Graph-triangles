#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <sys/time.h>
#include "readCSV.h"


double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void mulSparse(cooFormat* A, cooFormat* C, int N){

    // Initialize cuSPARSE
    hipsparseHandle_t handle;   
    hipsparseCreate(&handle);

    int nnzA = A->nnz;

    float* devVal;
    int* devCol, *devRow;
    hipMallocManaged(&devVal,nnzA*sizeof(float));
    hipMallocManaged(&devCol,nnzA*sizeof(int));
    hipMallocManaged(&devRow,nnzA*sizeof(int));
    hipMemcpy(devVal,A->cooValA,nnzA*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(devCol,A->cooColIndA,nnzA*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devRow,A->cooRowIndA,nnzA*sizeof(int),hipMemcpyHostToDevice);

    int* csrRowPtrA;
    hipMallocManaged(&csrRowPtrA,(N+1)*sizeof(int));
    // hipMemcpy(csrRowPtrA,A->cooRowIndA,(N+1)*sizeof(int),hipMemcpyHostToDevice);
    hipsparseXcoo2csr(handle,devRow,nnzA,N,csrRowPtrA,HIPSPARSE_INDEX_BASE_ZERO);
    

    // for (int i=0;i<N+1;i++){
    //     printf("csrValA from func: %f\n",devVal[i]);
    //     printf("csrRowPtrA from func: %d\n",csrRowPtrA[i]);
    //     printf("csrColIndA from func: %d\n",devCol[i]);
    // }
    // for (int i=0;i<nnzA;i++){
    //     printf("csrColIndA from func: %d for value: %f\n",devCol[i],devVal[i]);
    // }

    // printf("Inside func: %f\n",A->csrVal[4]);

    // Descriptor for sparse matrix A and C
    hipsparseMatDescr_t descrA,descrC;     
    hipsparseCreateMatDescr(&descrA);
    hipsparseCreateMatDescr(&descrC);

    int* d_C_RowPtr;
    hipMallocManaged(&d_C_RowPtr,(N+1)*sizeof(*d_C_RowPtr));

    int nnzC;
    //hipMallocManaged(&nnzC,sizeof(int));
    hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseXcsrgemmNnz(handle,transA,transA,N,N,N,descrA,nnzA,csrRowPtrA,devCol,descrA,nnzA,csrRowPtrA,devCol,descrC,d_C_RowPtr,&nnzC);


    printf("Non zero of C: %d\n",nnzC);

    float* d_C;
    int *d_C_ColIndices;
    hipMallocManaged(&d_C, (nnzC)*sizeof(*d_C));
    hipMallocManaged(&d_C_ColIndices, (nnzC) * sizeof(*d_C_ColIndices));
    double start = cpuSecond();
    hipsparseScsrgemm(handle,transA,transA,N,N,N,descrA,nnzA,devVal,csrRowPtrA,devCol,descrA,nnzA,devVal,csrRowPtrA,devCol,descrC,d_C,d_C_RowPtr,d_C_ColIndices);
    printf("Time elapsed for multiplication: %f seconds\n",cpuSecond()-start);

    int* cooRowC;
    hipMallocManaged(&cooRowC,(nnzC)*sizeof(int));

    hipsparseXcsr2coo(handle,d_C_RowPtr,(nnzC),N,cooRowC,HIPSPARSE_INDEX_BASE_ZERO);

    C->nnz = nnzC;
    C->cooValA = d_C;
    C->cooRowIndA = cooRowC;
    C->cooColIndA =  d_C_ColIndices;
}