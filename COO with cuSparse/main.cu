/**********************************************************************
 *
 * readCSV.c -- readCSV function for reading the input from
 *              a .csv file
 *
 * Michail Iason Pavlidis <michailpg@ece.auth.gr>
 * John Flionis <iflionis@auth.gr>
 *
 **********************************************************************/

 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include <ctype.h>  
 #include <inttypes.h>
 #include <errno.h>
 #include <math.h>
 #include "readCSV.h"

 

int main(int argc, char** argv){
 
    char* fName = argv[1];
    int N, M, nT_Mat;
    double matlab_time;

    cooFormat A, C;

    readCSV(fName, &A, &N, &M, &nT_Mat, &matlab_time);

    
    printf("Nonzeros = %d\n", A.nnz);
    

    printf("Validation File:\n N = %d, M = %d\n Matlab result was %d, produced in %lf\n", N, M, nT_Mat, matlab_time);

    mulSparse(&A,&C,N);

    float* devVal;
    int* devCol, *devRow;
    int nnzA = A.nnz;
    hipMallocManaged(&devVal,nnzA*sizeof(float));
    hipMallocManaged(&devCol,nnzA*sizeof(int));
    hipMallocManaged(&devRow,nnzA*sizeof(int));
    hipMemcpy(devVal,A.cooValA,nnzA*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(devCol,A.cooColIndA,nnzA*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(devRow,A.cooRowIndA,nnzA*sizeof(int),hipMemcpyHostToDevice);

    cooFormat B;
    B.cooColIndA = devCol;
    B.cooRowIndA = devRow;
    B.cooValA = devVal;
    B.nnz = A.nnz;

    int* sum, *counter, *counter1, *counter2;
    hipMallocManaged(&sum,sizeof(int));
    hipMallocManaged((void**)&counter,sizeof(int));
    hipMallocManaged(&counter1,sizeof(int));
    hipMallocManaged(&counter2,sizeof(int));

    *sum = 0;
    double st1 = cpuSecond();
    //filter<<<160,1024>>>(B,C,sum,counter2);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    //printf("Time filtering on GPU: %lf sec\n",cpuSecond()-st1);
    // printf("Vrhka tosa simeia: %d kai midenisa tosa: %d\n",*counter1,*counter2);

    double st2 = cpuSecond();
    //findTrianglesShared<<<160,1024>>>(B,C,sum,counter);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    //printf("Time on GPU using shared memory: %lf sec\n",cpuSecond()-st2);

    double st3 = cpuSecond();
    
    *counter = 0;
    *sum = 0;
    findTriangles<<<160,1024>>>(B,C,sum,counter);
    CHECK(hipPeekAtLastError());
    CHECK(hipDeviceSynchronize());
    printf("Triangles using COO format: %d\n",sum[0]/6);
    // printf("Triangles naive: %d\n",counter[0]/6);
    printf("Time on GPU using COO format: %lf sec\n",cpuSecond()-st3);
    
    double st = cpuSecond();
    // findTrianglesCPU(&B,&C);
    // printf("Time on CPU: %lf sec\n",cpuSecond()-st);


    
    return 0; 
}